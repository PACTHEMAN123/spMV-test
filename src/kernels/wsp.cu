#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include "wsp.hpp"

__global__ void wsp_kernel(
    int M, int N,
    int nz_max_m,
    uint32_t *bitmaps,
    float* A_vals,
    float *X, float *Y
) {
    int lane_id = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;
    int cur_col = blockIdx.x * 4 + warp_id;

    uint32_t curr_mask = (1u << lane_id);
    uint32_t prev_mask = (lane_id == 0) ? 0 : ((1u << lane_id) - 1);

    int A_val_row_cnt = 0;

    float sum = 0;

    // outer loop: each loop, 32 block
    for (int out_bk = 0; out_bk < M; out_bk += 32 * 32) {
        // for each warp, load 32 bitmaps for a big loop
        int bmp_start = cur_col * (M / 32) + out_bk / 32;
        uint32_t bitmap = bitmaps[bmp_start + lane_id];
        int nz_num = __popc(bitmap);

        // inner loop: each loop, 32 threads finish 1 block
        for (int i = 0; i < 32; i++) {
            uint32_t cur_bitmap = __shfl_sync(0xffffffff, bitmap, i);
            
            if (cur_bitmap & curr_mask) {
                int x_start = out_bk + i * 32;
                float x = X[x_start + lane_id];

                int A_val_in_blk_offset = __popc(cur_bitmap & prev_mask);
                int a_start = nz_max_m * cur_col + A_val_row_cnt;
                float a = A_vals[a_start + A_val_in_blk_offset];

                sum += x * a;
            }

            // update nz values counter
            int cur_nz_num = __shfl_sync(0xffffffff, nz_num, i);
            A_val_row_cnt += cur_nz_num;
        }
    }

    for (int i = 16; i >= 1; i >>= 1) {
        sum += __shfl_xor_sync(0xffffffff, sum, i);
    }

    if (lane_id == 0)
        Y[cur_col] = sum;
}

void wsp_gemv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    dim3 block(128, 1, 1);
    dim3 grid(N / 4, 1, 1);

    WSPMatrix wsp(M, N, A_host);

    const size_t size_A_val = wsp.ValuesSize() * sizeof(float);
    const size_t size_A_bmp = wsp.BitmapsSize() * sizeof(uint32_t);

    float *A_val_dev;
    uint32_t *A_bmp_dev;
    CUDA_CHECK(hipMalloc((void **)&A_val_dev, size_A_val));
    CUDA_CHECK(hipMalloc((void **)&A_bmp_dev, size_A_bmp));

    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_val_dev, wsp.GetValues(), size_A_val, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(A_bmp_dev, wsp.GetBitmaps(), size_A_bmp, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    TIME_KERNEL((wsp_kernel<<<grid, block>>>(
        M, N, 
        wsp.nz_max_m,
        A_bmp_dev,
        A_val_dev,
        X_device, Y_device
    )));

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(A_val_dev));
    CUDA_CHECK(hipFree(A_bmp_dev));
    CUDA_CHECK(hipFree(X_device));
    CUDA_CHECK(hipFree(Y_device));
    return;
}

