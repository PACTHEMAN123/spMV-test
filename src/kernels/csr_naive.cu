#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include "matrix_csr.hpp"


// using CSR format, naive
__global__ void csr_naive_kernel(
    int M, int N, 
    float *A_vals, int A_vals_size, 
    int *A_col_idxs, int A_col_idxs_size,
    int *A_row_ptrs, int A_row_ptrs_size,
    float *X, float *Y
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int begin = A_row_ptrs[idx];
    int end = (idx == N - 1) ? A_col_idxs_size : A_row_ptrs[idx + 1];
    
    float acc = 0.0f;
    for (int i = begin; i < end; i++) {
        acc += X[A_col_idxs[i]] * A_vals[i];
    }

    Y[idx] = acc;
}


void csr_naive_gemv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    dim3 block(32, 1, 1);
    dim3 grid(N / 32);

    CSRMatrix csr(M, N, A_host);

    const size_t size_A_row_ptrs = csr.RowPtrsSize() * sizeof(int);
    const size_t size_A_col_idxs = csr.ColIdxsSize() * sizeof(int);
    const size_t size_A_values = csr.ValuesSize() * sizeof(float);

    float *A_values_device; 
    int *A_col_idxs_device, *A_row_ptrs_device;
    CUDA_CHECK(hipMalloc((void **)&A_values_device, size_A_values));
    CUDA_CHECK(hipMalloc((void **)&A_col_idxs_device, size_A_col_idxs));
    CUDA_CHECK(hipMalloc((void **)&A_row_ptrs_device, size_A_row_ptrs));

    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_values_device, csr.GetValues(), size_A_values, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(A_col_idxs_device, csr.GetColIdxs(), size_A_col_idxs, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(A_row_ptrs_device, csr.GetRowPtrs(), size_A_row_ptrs, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    // call the kernel
    TIME_KERNEL((csr_naive_kernel<<<grid, block>>>(
        M, N, 
        A_values_device, csr.ValuesSize(),
        A_col_idxs_device, csr.ColIdxsSize(),
        A_row_ptrs_device, csr.RowPtrsSize(),
        X_device, Y_device
    )));
    
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(A_values_device));
    CUDA_CHECK(hipFree(A_col_idxs_device));
    CUDA_CHECK(hipFree(A_row_ptrs_device));
    CUDA_CHECK(hipFree(X_device));
    CUDA_CHECK(hipFree(Y_device));

    return;
}