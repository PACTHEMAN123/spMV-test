#include "hip/hip_runtime.h"

#include "kernel.hpp"

__global__ void naive_kernel(int M, int N, float *A, float *X, float *Y) {

    __shared__ float ldg_x_buffer[32];
    __shared__ float ldg_a_buffer[32][32];
    float acc = 0.0f;

    #pragma unroll
    for (int block_ks = 0; block_ks < M; block_ks += 32) {
        // load the x
        ldg_x_buffer[threadIdx.x] = X[block_ks + threadIdx.x];

        // load the A
        #pragma unroll
        for (int i = 0; i < 32; i++) {
            int ay = block_ks + i;
            int ax = blockIdx.x * blockDim.x + threadIdx.x;
            ldg_a_buffer[i][threadIdx.x] = A[ay * N + ax];
        }

        __syncthreads();

        // compute
        for (int i = 0; i < 32; i++) {
            acc += ldg_x_buffer[i] * ldg_a_buffer[i][threadIdx.x];
        }
    }

    Y[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

void spmv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    dim3 block(32, 1, 1);
    dim3 grid(N / 32);

    const size_t size_A = M * N * sizeof(float);
    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *A_device, *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&A_device, size_A));
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_device, A_host, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    // call the kernel
    naive_kernel<<<grid, block>>>(M, N, A_device, X_device, Y_device);

    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost);

    return;
}