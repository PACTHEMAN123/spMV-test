#include "hip/hip_runtime.h"

#include "kernel.hpp"

// version -1: cublas baseline
void cublas_gemv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    const size_t size_A = M * N * sizeof(float);
    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *A_device, *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&A_device, size_A));
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_device, A_host, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    // create a cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // resd = matd * vecd
    // hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                        //    int m, int n,
                        //    const float           *alpha,
                        //    const float           *A, int lda,
                        //    const float           *x, int incx,
                        //    const float           *beta,
                        //    float           *y, int incy)
    float alpha = 1.0f;
    float beta = 0.0f;
    TIME_KERNEL(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, A_device, N, X_device, 1, &beta, Y_device, 1));

    hipblasDestroy(handle);

    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost);

    return;
}

// version 0: naive
__global__ void naive_kernel(int M, int N, float *A, float *X, float *Y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float acc = 0.0f;
    for (int i = 0; i < M; i++) {
        acc += X[i] * A[i * N + idx];
    }
    Y[idx] = acc;
}

void naive_gemv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    dim3 block(32, 1, 1);
    dim3 grid(N / 32);

    const size_t size_A = M * N * sizeof(float);
    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *A_device, *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&A_device, size_A));
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_device, A_host, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    // call the kernel
    TIME_KERNEL((naive_kernel<<<grid, block>>>(M, N, A_device, X_device, Y_device)));
    
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost);

    return;
}


// version 1: tiling + share memory
__global__ void tiling_kernel(int M, int N, float *A, float *X, float *Y) {

    __shared__ float ldg_x_buffer[32];
    __shared__ float ldg_a_buffer[32][32];
    float acc = 0.0f;

    #pragma unroll
    for (int block_ks = 0; block_ks < M; block_ks += 32) {
        // load the x
        ldg_x_buffer[threadIdx.x] = X[block_ks + threadIdx.x];

        // load the A
        #pragma unroll
        for (int i = 0; i < 32; i++) {
            int ay = block_ks + i;
            int ax = blockIdx.x * blockDim.x + threadIdx.x;
            ldg_a_buffer[i][threadIdx.x] = A[ay * N + ax];
        }

        __syncthreads();

        // compute
        for (int i = 0; i < 32; i++) {
            acc += ldg_x_buffer[i] * ldg_a_buffer[i][threadIdx.x];
        }
    }

    Y[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

void tiling_gemv_gpu(int M, int N, float *A_host, float *X_host, float *Y_host) {
    dim3 block(32, 1, 1);
    dim3 grid(N / 32);

    const size_t size_A = M * N * sizeof(float);
    const size_t size_X = M * sizeof(float);
    const size_t size_Y = N * sizeof(float);

    float *A_device, *X_device, *Y_device;
    CUDA_CHECK(hipMalloc((void **)&A_device, size_A));
    CUDA_CHECK(hipMalloc((void **)&X_device, size_X));
    CUDA_CHECK(hipMalloc((void **)&Y_device, size_Y));

    CUDA_CHECK(hipMemcpy(A_device, A_host, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(X_device, X_host, size_X, hipMemcpyHostToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    // call the kernel
    TIME_KERNEL((tiling_kernel<<<grid, block>>>(M, N, A_device, X_device, Y_device)));
    
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(Y_host, Y_device, size_Y, hipMemcpyDeviceToHost);

    return;
}

